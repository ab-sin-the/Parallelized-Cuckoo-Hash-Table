#include <iostream>
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <ctime>

using namespace std;

__device__ int hashing_d (int element, int a, int b, int c, int p, int n){
	return (unsigned int)(a * element + b) % p % n;
}


int hashing (int element, int a, int b, int c, int p, int n){
	return (unsigned int)(a * element + b) % p % n;
}

__global__ void add_two(int p, int n, int N, int t, int* hash_table, int* hash_table2, int* hash_elements, int* func_table, int* a, int* b, int* c){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int func1 = 0;
	int func2 = 1;
	int func3 = 2;
	int hash_element = hash_elements[i];
	unsigned int loca1 = hashing_d(hash_element, a[func1], b[func1], c[func1], p, n);
	unsigned int loca2 = hashing_d(hash_element, a[func2], b[func2], c[func2], p, n);
	unsigned int loca3 = hashing_d(hash_element, a[func3], b[func3], c[func3], p, n);
	atomicAdd(&hash_table2[(unsigned int)loca1], 1);
	atomicAdd(&hash_table2[(unsigned int)loca2], 1);
	atomicAdd(&hash_table2[(unsigned int)loca3], 1);
}

__global__ void fix_location(int p, int n, int N, int t, int* hash_table, int* hash_table2, int* hash_elements, int* func_table, int* a, int* b, int* c){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int func1 = 0;
	int func2 = 1;
	int func3 = 2;
	int hash_element = hash_elements[i];
	unsigned int loca1 = hashing_d(hash_element, a[func1], b[func1], c[func1], p, n);
	unsigned int loca2 = hashing_d(hash_element, a[func2], b[func2], c[func2], p, n);
	unsigned int loca3 = hashing_d(hash_element, a[func3], b[func3], c[func3], p, n);
	if (hash_table2[(unsigned int)loca1] == 1){
		func_table[i] = 0;
	}
	if (hash_table2[(unsigned int)loca2] == 1){
		func_table[i] = 1;
	}
	if (hash_table2[(unsigned int)loca3] == 1){
		func_table[i] = 2;
	}
}

__global__ void check(int p, int n, int N, int t, int* hash_table, int* hash_elements, int* func_table, int* a, int* b, int* c, int max_count, int *indicator){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int curr_func = func_table[i];
	int hash_element = hash_elements[i];
	unsigned int loca = hashing_d(hash_element, a[curr_func], b[curr_func], c[curr_func], p, n);
	if (hash_table[(unsigned int)loca] != hash_element){
		*indicator = -1;
	}
}

__global__ void insert(int p, int n, int N, int t, int* hash_table, int* hash_elements, int* func_table, int* a, int* b, int* c, int max_count){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < N){
		int curr_func = func_table[i];
		int hash_element = hash_elements[i];
		unsigned int loca = hashing_d(hash_element, a[curr_func], b[curr_func], c[curr_func], p, n);

		//hash_table[(unsigned int)loca] = hash_element;
		if (hash_table[(unsigned int)loca] != hash_element){
			func_table[i] = (curr_func + 1) % t;
			curr_func = (curr_func + 1) % t;
			loca = hashing_d(hash_element, a[curr_func], b[curr_func], c[curr_func], p, n);
			hash_table[(unsigned int)loca] = hash_element;
		}
	/*for (int j = 0; j < max_count; j++){
		if (hash_table[(unsigned int)loca] != hash_element){
			func_table[i] = (curr_func + 1) % t;
			curr_func = (curr_func + 1) % t;
			loca = ((unsigned int)(a[curr_func] * hash_element + b[curr_func]) % p % n);
			hash_table[(unsigned int)loca] = hash_element;
		}
		__syncthreads();
	}*/

	}

}




__global__ void find(int p, int n, int N, int t, int* hash_table, int* find_elements, int* a, int* b, int* c, int* find_result){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int find = 0;
	for (int j = 0; j < t; j++){
		if (hash_table[hashing_d(find_elements[i], a[j], b[j], c[j], p, n)] == find_elements[i]){
			find = 1;
		}
	}
	find_result[i] = find;
}



__global__ void delete_ele(int p, int n, int N, int t, int* hash_table, int* fun_index_table,int* delete_elements, int* a, int* b, int* c, int* find_result){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	for (int j = 0; j < t; j++){
		if (hash_table[hashing_d(delete_elements[i], a[j], b[j], c[j], p, n)] == delete_elements[i]){
			hash_table[hashing_d(delete_elements[i], a[j], b[j], c[j], p, n)] = 0;
			fun_index_table[hashing_d(delete_elements[i], a[j], b[j], c[j], p, n)] = t;
		}
	}
}

void random_hash_fun(int t, int* a, int* b, int* c){
	for (int i = 0; i < t; i++){
		a[i] = rand();
		b[i] = rand();
		c[i] = rand();
	}
	return;
}

void random_hash_elements(int N, int* hash_elements){
	for (int i = 0; i < N; i++){
		hash_elements[i] = rand() % (1 << 27);
	}
	return;
}

void initialize(int n, int N, int* hash_table, int* hash_table2, int* func_table, int* hash_value, int* hash_elements, int a, int b, int c, int p ){
	for (int i = 0; i < n; i++){
		hash_table[i] = 0;
		hash_table2[i] = 0;
	}
	for (int i = 0; i < N; i++){
		int loca = hashing(hash_elements[i], a, b, c, p, n);
		hash_value[i] = loca;
		func_table[i] = 0;
	}
}



void generating_find_element(int N, int* find_elements, int* hash_element, float partial){
	for (int i = 0; i < (int) (N * partial); i++){
		find_elements[i] = hash_element[i];
	}

	for (int i = (int) (N * partial); i < N; i++){
		find_elements[i] = rand() % (1 << 27);
	}
}

void quicksort(int * hash_value, int* hash_elements, int low, int high)
{
    if(low >= high){
        return;
    }
    int first = low;
    int last = high;
    int key = hash_value[first];
    int key1 = hash_elements[first];
    while(first < last){
        while(first < last && hash_value[last] >= key){
        	--last;
        }

        hash_value[first] = hash_value[last];
        hash_elements[first] = hash_elements[last];
 
        while(first < last && hash_value[first] <= key){
            ++first;
        }
        hash_value[last] = hash_value[first]; 
        hash_elements[last] = hash_elements[first];    
    }
    hash_value[first] = key;
    hash_elements[first] = key1;
    quicksort(hash_value, hash_elements, low, first-1);
    quicksort(hash_value, hash_elements, first+1, high);
}


double once(int p, int n, int t, int N, int max_count, int trail, float partial, int thread_per_block){
	cout << "Trail: " << trail << endl;
	int* hash_table;
	int* hash_table2;
	int* hash_elements;
	int* func_table;
	int* a;
	int* b;
	int* c;
	int* find_result; 
	int* find_elements; 
	int* indicator;
	int count = 0;
	int* hash_value = new int [N];

	hipMallocManaged(&hash_table, n * sizeof(int));
	hipMallocManaged(&hash_table2, n * sizeof(int));
	hipMallocManaged(&hash_elements, N * sizeof(int));
	hipMallocManaged(&func_table, N * sizeof(int));
	hipMallocManaged(&a, t * sizeof(int));
	hipMallocManaged(&b, t * sizeof(int));
	hipMallocManaged(&c, t * sizeof(int));
	hipMallocManaged(&find_elements, N * sizeof(int));
	hipMallocManaged(&find_result, N * sizeof(int));
	hipMallocManaged(&indicator, sizeof(int));

	*indicator = 0;


	random_hash_fun(t, a, b, c);
	random_hash_elements(N, hash_elements);
	initialize(n, N, hash_table, hash_table2, func_table, hash_value, hash_elements, a[0], b[0], c[0], p);


	quicksort(hash_value, hash_elements, 0, N - 1);

	add_two<<< ceil(N / thread_per_block), thread_per_block>>>(p, n, N, t, hash_table, hash_table2, hash_elements, func_table, a, b, c);
	hipDeviceSynchronize();
	fix_location<<< ceil(N / thread_per_block), thread_per_block>>>(p, n, N, t, hash_table, hash_table2, hash_elements, func_table, a, b, c);
	hipDeviceSynchronize();


	for (count = 0; count < max_count; count ++){
		*indicator = 0;
		insert<<< ceil(N / thread_per_block), thread_per_block>>>(p, n, N, t, hash_table, hash_elements, func_table, a, b, c, max_count);
		hipDeviceSynchronize();
		check<<< ceil(N / thread_per_block), thread_per_block>>>(p, n, N, t, hash_table, hash_elements, func_table, a, b, c, max_count, indicator);
		hipDeviceSynchronize();
		if (*indicator == 0){
			break;
		}
	}

	check<<< ceil(N / thread_per_block), thread_per_block>>>(p, n, N, t, hash_table, hash_elements, func_table, a, b, c, max_count, indicator);
	hipDeviceSynchronize();

	int count1 = 0;
	while (*indicator == -1 && count1 < 1000){
		random_hash_fun(t, a, b, c);
		add_two<<< ceil(N / thread_per_block), thread_per_block>>>(p, n, N, t, hash_table, hash_table2, hash_elements, func_table, a, b, c);
		hipDeviceSynchronize();
		fix_location<<< ceil(N / thread_per_block), thread_per_block>>>(p, n, N, t, hash_table, hash_table2, hash_elements, func_table, a, b, c);
		hipDeviceSynchronize();

		for (count = 0; count < max_count; count ++){
			*indicator = 0;
			insert<<< ceil(N / thread_per_block), thread_per_block>>>(p, n, N, t, hash_table, hash_elements, func_table, a, b, c, max_count);
			hipDeviceSynchronize();
			check<<< ceil(N / thread_per_block), thread_per_block>>>(p, n, N, t, hash_table, hash_elements, func_table, a, b, c, max_count, indicator);
			hipDeviceSynchronize();
			if (*indicator == 0){
				break;
			}
		}
		check<<< ceil(N / thread_per_block), thread_per_block>>>(p, n, N, t, hash_table, hash_elements, func_table, a, b, c, max_count, indicator);
		hipDeviceSynchronize();
		count1 ++;
	}

	generating_find_element(N, find_elements, hash_elements, partial);

	clock_t start = clock();
	find<<< ceil(N / thread_per_block), thread_per_block>>>(p, n, N, t, hash_table, find_elements, a, b, c, find_result);
	
	hipDeviceSynchronize();

	double duration = (clock() - start) / (double) CLOCKS_PER_SEC;


	cout << "Time:" << duration << endl;
	hipMemcpy(hash_table, hash_table, n * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hash_elements, hash_elements, N * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(find_result, find_result, N * sizeof(int), hipMemcpyDeviceToHost);

	int val_count = 0;

	val_count = 0;
	for (int i = 0 ; i < N; i++){
		if(find_result[i] == 1){
			val_count ++;
		}
	}

	cout << "Difference between insertion and find result: " << N - val_count << endl;


	hipFree(hash_table);
	hipFree(hash_elements);
	hipFree(find_result);
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(hash_table2);
	hipFree(func_table);
	hipFree(indicator);
	return duration;
}

int main(void){
	int p = 99984923;
	int count_bit;
	float relative;
	float partial;
	int n;
	int c;
	cout << "Data Bit: " ;
	cin >> count_bit;
	cout << "Find partial: ";
	cin >> partial;
	cout << "Hash table size (0 for 2^25): ";
	cin >> relative; 
	cout << "Evict Chain Constant: ";
	cin >> c;
	int N = (1<<count_bit); 

	if (relative == 0){
		n = 1 << 25;
	}else{
		n = relative * N;
	}

	int t = 3;
	int max_count = (int) c * log(n);
	int thread_per_block;
	if (count_bit >= 18){
		thread_per_block = 1024;
	}else{
		thread_per_block = 64;
	}
	double Time = 0;

	Time += once(p, n, t, N, max_count, 1, partial, thread_per_block);
	Time += once(p, n, t, N, max_count, 2, partial, thread_per_block);
	Time += once(p, n, t, N, max_count, 3, partial, thread_per_block);
	Time += once(p, n, t, N, max_count, 4, partial, thread_per_block);
	Time += once(p, n, t, N, max_count, 5, partial, thread_per_block);

	Time = Time / 5;
	cout << "Average Time: " << Time << endl; 
	cout << "Million of insertion per second: " << (N / Time) / 1000000 << endl;
	return 0;
}
